#include "hip/hip_runtime.h"

#include "Kernel.cuh"


__global__ void kernel(State* initialState, State* returnedPath, int PATHMAXSIZE, double rand){
	hiprandState_t hiprandState;
	/* we have to initialize the state */
	hiprand_init(clock(), /* the seed controls the sequence of random values that are produced */
		0, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&hiprandState);

	//returnedPath[0] = State((hiprand_uniform(&hiprandState) * 2) - 1, (hiprand_uniform(&hiprandState) * 4 * 3.14f)  - 1.57f, initialState);
	printf("mthfk\n");
}

std::vector<State> cuda_search(State initialState){

	const int PATHMAXSIZE = 5;

	State* auxInitState;

	State* auxReturnedPath;
	
	State returnedPath[1];


	hipMalloc(&auxInitState, sizeof(State));
	hipMemcpy(auxInitState, &initialState, sizeof(State), hipMemcpyHostToDevice);

	hipMalloc(&auxReturnedPath, sizeof(State)*PATHMAXSIZE);
	hipMemcpy(auxReturnedPath, &returnedPath, sizeof(State)*PATHMAXSIZE, hipMemcpyHostToDevice);

	//kernel << < 64, 64 >> > (auxInitState, auxReturnedPath, PATHMAXSIZE);
	//srand(time(NULL));
	//double rand = (std::rand() / ((double)RAND_MAX / 2)) - 1;
	kernel << < 1, 1 >> > (auxInitState, auxReturnedPath, PATHMAXSIZE, 0);
	hipMemcpy(&returnedPath, auxReturnedPath, sizeof(State)*PATHMAXSIZE, hipMemcpyDeviceToHost);

	hipFree(auxInitState);
	hipFree(auxReturnedPath);

	/*hipDeviceSynchronize();
	std::cout << "error: " << hipGetErrorString(hipPeekAtLastError()) << std::endl;
*/

	std::vector<State> ret = std::vector<State>(1);
	//ret.assign(returnedPath, returnedPath + sizeof(returnedPath)); //transform array in vector

	return ret;
}
