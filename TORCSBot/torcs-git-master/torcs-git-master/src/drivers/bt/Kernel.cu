#include "hip/hip_runtime.h"

#include "Kernel.cuh"


__global__ void CUDAProcedure(tTrackSeg* segArray, int nTrackSegs, State* graph, int stateIterator, 
	double minXVertex, double maxXVertex, double minYVertex, double maxYVertex, 
	int numThreads, double maxPathCost, State* bestPath,
	int forwardSegments, double neighborDeltaPos, double neighborDeltaSpeed, double actionSimDeltaTime){

	int idx = threadIdx.x + blockDim.x*blockIdx.x;

	hiprandState_t hiprandState;


	/* we have to initialize the state */
	hiprand_init(clock(), /* the seed controls the sequence of random values that are produced */
		idx, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&hiprandState);

	int offset = stateIterator*numThreads + idx;

	double trackMapXMin = minXVertex;
	double trackMapXMax = maxXVertex;

	double trackMapXDelta = trackMapXMax - trackMapXMin;

	double trackMapYMin = minYVertex;
	double trackMapYMax = maxYVertex;

	double trackMapYDelta = trackMapYMax - trackMapYMin;


	double trackMapZMin = 0;
	double trackMapZMax = 20;

	double trackMapZDelta = trackMapZMax - trackMapZMin;

	double minSpeed = 0;
	double maxSpeed = 60;

	double speedDelta = maxSpeed - minSpeed;


	double minAccel = 0;
	double maxAccel = 10;

	double accelDelta = maxAccel - minAccel;


	//------------------------generate random point --------------------------------

	double randPosX = trackMapXDelta * hiprand_uniform(&hiprandState) + trackMapXMin;
	double randPosY = trackMapYDelta * hiprand_uniform(&hiprandState) + trackMapYMin;
	double randPosZ = trackMapZDelta * hiprand_uniform(&hiprandState) + trackMapZMin;

	tPosd randPos;
	randPos.x = randPosX;
	randPos.y = randPosY;
	randPos.z = randPosZ;

	double randSpeedX = speedDelta * hiprand_uniform(&hiprandState) + minSpeed;
	double randSpeedY = speedDelta * hiprand_uniform(&hiprandState) + minSpeed;
	double randSpeedZ = speedDelta * hiprand_uniform(&hiprandState) + minSpeed;

	tPosd randSpeed;
	randSpeed.x = randSpeedX;
	randSpeed.y = randSpeedY;
	randSpeed.z = randSpeedZ;

	double randAccelX = accelDelta * hiprand_uniform(&hiprandState) + minAccel;
	double randAccelY = accelDelta * hiprand_uniform(&hiprandState) + minAccel;
	double randAccelZ = accelDelta * hiprand_uniform(&hiprandState) + minAccel;

	tPosd randAccel;
	randAccel.x = randAccelX;
	randAccel.y = randAccelY;
	randAccel.z = randAccelZ;

	State xRand = State(randPos, randSpeed, randAccel);

	//------------------------------find parent--------------------------------------

	//the generation didnt work
	if (!ConstraintChecking::validPoint(segArray, nTrackSegs, &xRand, (double) -2.0)){
		return;
	}
		
	State* xNearest = Kernel::nearestNeighbor(&xRand, graph, 200, actionSimDeltaTime); //GRAPH ITERATOR FUCKUP!
	xRand.setParent(xNearest);

	//------------------------------apply delta--------------------------------------

	DeltaHeuristics::applyDelta(&xRand, xNearest, segArray, nTrackSegs, forwardSegments, neighborDeltaPos, neighborDeltaSpeed);


	double cMin = xNearest->getPathCost() + EvalFunctions::evaluatePathCost(segArray, nTrackSegs, xNearest, &xRand, forwardSegments); //redifine path cost for new coords
	xRand.setPathCost(cMin);

	//printf("parent out!:%f:%f\n", xRand.getPos().x, xRand.getParent()->getPos().x);

	//the delta application didnt work
	if (!ConstraintChecking::validPoint(segArray, nTrackSegs, &xRand, (double) -2.0)){
		return;
	}


	//------------------------------push to graph--------------------------------------
	graph[offset] = xRand;

	//------------------------------check best path--------------------------------------

	if (xRand.getPathCost() >= maxPathCost){ //reevaluate best path

		maxPathCost = xRand.getPathCost();
		delete bestPath;
		int pathIndex = 0;
		State* initialNode = &graph[0];
		while (!xRand.getInitialState()){
			//printf("backtracking:%f\n", bestState->getPos().x);
			bestPath[pathIndex] = xRand;
			xRand = *xRand.getParent();
			pathIndex++;
		}
		bestPath[pathIndex] = *initialNode;
	}
	
}


State* Kernel::callKernel(tTrackSeg* segArray, int nTrackSegs,State* initialState,
	double minXVertex, double maxXVertex, double minYVertex, double maxYVertex,
	double numIterations,
	int forwardSegments, double neighborDeltaPos, double neighborDeltaSpeed, double actionSimDeltaTime){

	State* bestPath = new State[(unsigned int)numIterations];
	State* auxBestPath;

	State* graph = new State[(unsigned int)numIterations];
	graph[0] = *initialState;


	State* auxGraph;
	tTrackSeg* auxSegArray;


	double maxPathCost = -1 * DBL_MAX; //force a change


	int NUM_BLOCKS = 3;
	int NUM_THREADS_EACH_BLOCK = 5;
	int NUM_THREADS = NUM_BLOCKS*NUM_THREADS_EACH_BLOCK;

	int numPartialIterations = numIterations / NUM_THREADS;

	if (numPartialIterations == 0) numPartialIterations++;


	hipMalloc(&auxGraph, sizeof(State)*(unsigned int)numIterations);
	hipMalloc(&auxSegArray, sizeof(tTrackSeg)*(unsigned int)nTrackSegs);
	hipMalloc(&auxBestPath, sizeof(State)*(unsigned int)numIterations);

	hipMemcpy(auxBestPath, bestPath, sizeof(State)*(unsigned int)numIterations, hipMemcpyHostToDevice);
	hipMemcpy(auxSegArray, segArray, sizeof(tTrackSeg)*(unsigned int)nTrackSegs, hipMemcpyHostToDevice);
	hipMemcpy(auxGraph, graph, sizeof(State)*(unsigned int)numIterations, hipMemcpyHostToDevice);

	for (int i = 0; i < numPartialIterations; i++)
	{
		CUDAProcedure << < NUM_BLOCKS, NUM_THREADS_EACH_BLOCK >> > (auxSegArray, nTrackSegs, auxGraph, i,
			minXVertex, maxXVertex, minYVertex, maxYVertex,
			NUM_THREADS, maxPathCost, auxBestPath,
			forwardSegments, neighborDeltaPos, neighborDeltaSpeed, actionSimDeltaTime);

		hipDeviceSynchronize();
	}
	

	hipMemcpy(graph, auxGraph, sizeof(State)*(unsigned int)numIterations, hipMemcpyDeviceToHost);
	hipMemcpy(bestPath, auxBestPath, sizeof(State)*(unsigned int)numIterations, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	/*for (int i = 0; i < 10; i++){
		printf("dgraphPos!:%f\n", graph[i].getPos().x);
	}*/

	hipFree(auxGraph);
	hipFree(auxSegArray);
	hipFree(auxBestPath);

	std::cout << "gcnsjfddgjvhgdffgsgfuiserror: " << hipGetErrorString(hipPeekAtLastError()) << std::endl;


	delete[] graph;

	return bestPath;
}
