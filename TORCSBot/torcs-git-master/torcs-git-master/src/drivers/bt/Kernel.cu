#include "hip/hip_runtime.h"

#include "Kernel.cuh"


CUDA_GLOBAL
void warmStart(int* f)
{
	*f = 0;
}

CUDA_GLOBAL
void CUDAProcedure(tTrackSeg* trackSegArray, int nTrackSegs, State* graph, int stateIterator,
	int numThreads, int graphSize, double maxCost, double actionSimDeltaTime){

	int idx = threadIdx.x + blockDim.x*blockIdx.x;

	int offset = (stateIterator*numThreads + idx) + 1; //the initial state does not need this computation

	hiprandState_t hiprandState;


	/* we have to initialize the state */
	hiprand_init(clock(), /* the seed controls the sequence of random values that are produced */
		idx, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&hiprandState);


	State initialState = graph[0];

	State xRand;

	//--------------------------------------- generate random sample -----------------------------------------------

	xRand = RandomStateGenerators::uniformRandomState(trackSegArray, nTrackSegs, &hiprandState);
	//xRand = RandomStateGenerators::gaussianRandomState(trackSegArray, nTrackSegs, startSegIndex, finalIndex, initialState.getVelocity());


	//---------------------------------------- select neighbor ----------------------------------------------------

	State xNearest = UtilityMethods::nearestNeighbor(xRand, graph, graphSize);
	xRand.setParentGraphIndex(xNearest.getMyGraphIndex());
	xRand.setLevelFromStart(xNearest.getLevelFromStart() + 1);

	////----------------------------------------- constraint checking ------------------------------------------------

	////if the acceleration is too much for the car to handle, prune the state
	///*if (abs(xRand.getVelocity().angle- xNearest.getVelocity().angle) > maxCarAcceleration.angle || abs(xRand.getVelocity().intensity-xNearest.getVelocity().intensity) > maxCarAcceleration.intensity){
	//	return;
	//}*/

	//the delta application also checks if the trajectory is valid
	if (!DeltaFunctions::applyDelta(&xRand, &xNearest, trackSegArray, nTrackSegs, actionSimDeltaTime)){
		return;
	}

	xRand.setMyGraphIndex(offset);
	graph[offset]= xRand;


}

//This method takes off the CUDA initialization delay on first call during real-time
// because it is done during loading (pre-computing phase)
void Kernel::gpuWarmup(){
	int count;
	hipDeviceProp_t prop;

	hipGetDeviceCount(&count);

	// --- print devices info ----------------------------------------------------------

	for (int i = 0; i<count; i++) {

		hipGetDeviceProperties(&prop, i);

		printf("---General  Information for device %d---\n", i);
		printf("Name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %d\n", prop.clockRate);
		printf("Device copy overlap: ");
		if (prop.deviceOverlap) printf("Enabled\n");
		else printf("Disabled\n");
		printf("Kernel execution timeout: ");
		if (prop.kernelExecTimeoutEnabled) printf("Enabled\n");
		else printf("Disabled\n");

		printf("---  Memory Information for device %d ---\n", i);
		printf("Total global mem: %ld\n", prop.totalGlobalMem);
		printf("Total constant mem: %ld\n", prop.totalConstMem);
		printf("Max mem pitch: %ld\n", prop.memPitch);
		printf("Texture alignment: %ld\n", prop.textureAlignment);

		printf("--- MP Information for device %d ---\n", i);
		printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp: %d\n", prop.regsPerBlock);
		printf("Threads in warp: %d\n", prop.warpSize);
		printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0],
			prop.maxThreadsDim[1],
			prop.maxThreadsDim[2]);
		printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0],
			prop.maxGridSize[1],
			prop.maxGridSize[2]);

		printf("\n");
	}
	
	// -- create a context -- kernel warmup ---------------------------------------------

	int *f = NULL;
	hipMalloc(&f, sizeof(int));
	warmStart <<< 1, 1 >> >(f);
	hipFree(f);

	hipDeviceSynchronize();
	printf( "kernel error : %s" , hipGetErrorString(hipPeekAtLastError()) );


	//-----------------------------------------------------------------------------------

}

State* Kernel::callKernel(tTrackSeg* segArray, int nTrackSegs, State* initialState, int numIterations, double actionSimDeltaTime){

	int graphSize = numIterations + 1;

	State* graph = new State[(unsigned int)graphSize];
	initialState->setMyGraphIndex(0);
	graph[0] = *initialState;


	State* auxGraph;
	tTrackSeg* auxSegArray;

	double maxPathCost = 0; //just to mock (was not removed as it can still be needed)

	int NUM_BLOCKS = 1;
	int NUM_THREADS_EACH_BLOCK = 100;
	int NUM_THREADS = NUM_BLOCKS*NUM_THREADS_EACH_BLOCK;

	float iterationRatio = (float) numIterations / (float) NUM_THREADS;
	int numPartialIterations = 0;
	numPartialIterations = ceilf(iterationRatio) == iterationRatio ? (int) iterationRatio : (int) iterationRatio + 1;
	
	if (numPartialIterations == 0) numPartialIterations++;


	clock_t mallocTimer;
	clock_t memcpyTimer1;
	clock_t kernelCallTimer;
	clock_t syncronizeTimer;
	clock_t memcpyTimer2;

	mallocTimer = clock();

	hipMalloc(&auxGraph, sizeof(State)*(unsigned int)graphSize);
	hipMalloc(&auxSegArray, sizeof(tTrackSeg)*(unsigned int)nTrackSegs);

	mallocTimer = clock() - mallocTimer;
	printf("malloc timer: %f \n" , double(mallocTimer) / (double)CLOCKS_PER_SEC );

	memcpyTimer1 = clock();

	//hipMemcpy(auxBestPath, bestPath, sizeof(State)*(unsigned int)numIterations, hipMemcpyHostToDevice);
	hipMemcpy(auxSegArray, segArray, sizeof(tTrackSeg)*(unsigned int)nTrackSegs, hipMemcpyHostToDevice);
	hipMemcpy(auxGraph, graph, sizeof(State)*(unsigned int)graphSize, hipMemcpyHostToDevice);

	memcpyTimer1 = clock() - memcpyTimer1;
	//std::cout << "memcpy1 timer: " << double(memcpyTimer1) / (double) CLOCKS_PER_SEC << std::endl;

	for (int i = 0; i < numPartialIterations; i++)
	{
		kernelCallTimer = clock();

		CUDAProcedure << < NUM_BLOCKS, NUM_THREADS_EACH_BLOCK >> > (auxSegArray, nTrackSegs, auxGraph, i,
			NUM_THREADS, graphSize, maxPathCost, actionSimDeltaTime);

		kernelCallTimer = clock() - kernelCallTimer;

		syncronizeTimer = clock();

		hipDeviceSynchronize();

		syncronizeTimer = clock() - syncronizeTimer;

		printf("kernell call timer: %f \n", double(kernelCallTimer) / (double)CLOCKS_PER_SEC);
		printf("sync timer: %f \n" ,double(syncronizeTimer) / (double)CLOCKS_PER_SEC );

	}
	
	memcpyTimer2 = clock();

	hipMemcpy(graph, auxGraph, sizeof(State)*(unsigned int)graphSize, hipMemcpyDeviceToHost);
	//hipMemcpy(bestState, auxBestState, sizeof(State)*(unsigned int)numIterations, hipMemcpyDeviceToHost);

	memcpyTimer2 = clock() - memcpyTimer2;

	printf("memcpyTimer2 timer: %f \n", double(memcpyTimer2) / (double)CLOCKS_PER_SEC);


	hipFree(auxGraph);
	hipFree(auxSegArray);

	hipDeviceSynchronize();
	printf("kernel error: %s" , hipGetErrorString(hipPeekAtLastError()) );


	

	return graph;
}
