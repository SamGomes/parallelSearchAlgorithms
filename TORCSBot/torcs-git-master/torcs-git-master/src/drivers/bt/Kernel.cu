#include "hip/hip_runtime.h"

#include "Kernel.cuh"


__global__ void kernel(State* initialState, State* returnedPath, int PATHMAXSIZE, double rand){
	hiprandState_t hiprandState;
	/* we have to initialize the state */
	hiprand_init(clock(), /* the seed controls the sequence of random values that are produced */
		0, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&hiprandState);
	double randInc = (double)hiprand_uniform(&hiprandState) * (double) 50;

	tPosd randPos = { initialState->getPos().x + randInc
		, initialState->getPos().y + randInc
		, initialState->getPos().z + randInc };
	
	tPosd randSpeed = { initialState->getSpeed().x + randInc
		, initialState->getSpeed().y + randInc
		, initialState->getSpeed().z + randInc };

	returnedPath[0] = State(randPos,randSpeed,initialState->getAcceleration(),initialState);
	//printf("executed Kernel!\n");
}

State* cuda_search(State initialState){

	const int PATHMAXSIZE = 1;

	State* auxInitState;

	State* auxReturnedPath;
	
	State returnedPath[1];


	hipMalloc(&auxInitState, sizeof(State));
	hipMemcpy(auxInitState, &initialState, sizeof(State), hipMemcpyHostToDevice);

	hipMalloc(&auxReturnedPath, sizeof(State)*PATHMAXSIZE);
	hipMemcpy(auxReturnedPath, &returnedPath, sizeof(State)*PATHMAXSIZE, hipMemcpyHostToDevice);

	kernel << < 10, 2 >> > (auxInitState, auxReturnedPath, PATHMAXSIZE,0);
	
	//kernel << < 1, 1 >> > (auxInitState, auxReturnedPath, PATHMAXSIZE, 0);
	hipMemcpy(&returnedPath, auxReturnedPath, sizeof(State)*PATHMAXSIZE, hipMemcpyDeviceToHost);
	
	hipDeviceSynchronize();
	
	hipFree(auxInitState);
	hipFree(auxReturnedPath);

	
	//std::cout << "error: " << hipGetErrorString(hipPeekAtLastError()) << std::endl;
	

	return new State(*returnedPath);
}
