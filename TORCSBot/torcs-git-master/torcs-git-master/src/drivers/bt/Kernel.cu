#include "hip/hip_runtime.h"

#include "Kernel.cuh"


__global__ void warmStart(int* f)
{
	*f = 0;
}

__global__ void CUDAProcedure(tTrackSeg* segArray, int nTrackSegs, State* graph, int stateIterator, 
	double minXVertex, double maxXVertex, double minYVertex, double maxYVertex, 
	int numThreads, double maxPathCost, State* bestState,
	int forwardSegments, double neighborDeltaPos, double neighborDeltaSpeed, double actionSimDeltaTime){

	int idx = threadIdx.x + blockDim.x*blockIdx.x;

	int offset = (stateIterator*numThreads + idx) + 1; //the initial state does not need this computation

	hiprandState_t hiprandState;


	/* we have to initialize the state */
	hiprand_init(clock(), /* the seed controls the sequence of random values that are produced */
		idx, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&hiprandState);


	double trackMapXMin = minXVertex;
	double trackMapXMax = maxXVertex;

	double trackMapXDelta = trackMapXMax - trackMapXMin;

	double trackMapYMin = minYVertex;
	double trackMapYMax = maxYVertex;

	double trackMapYDelta = trackMapYMax - trackMapYMin;


	double trackMapZMin = 0;
	double trackMapZMax = 20;

	double trackMapZDelta = trackMapZMax - trackMapZMin;

	double minSpeed = 0;
	double maxSpeed = 60;

	double speedDelta = maxSpeed - minSpeed;


	double minAccel = 0;
	double maxAccel = 10;

	double accelDelta = maxAccel - minAccel;


	//------------------------generate random point --------------------------------

	double randPosX = trackMapXDelta * hiprand_uniform(&hiprandState) + trackMapXMin;
	double randPosY = trackMapYDelta * hiprand_uniform(&hiprandState) + trackMapYMin;
	double randPosZ = trackMapZDelta * hiprand_uniform(&hiprandState) + trackMapZMin;

	tPosd randPos;
	randPos.x = randPosX;
	randPos.y = randPosY;
	randPos.z = randPosZ;

	double randSpeedX = speedDelta * hiprand_uniform(&hiprandState) + minSpeed;
	double randSpeedY = speedDelta * hiprand_uniform(&hiprandState) + minSpeed;
	double randSpeedZ = speedDelta * hiprand_uniform(&hiprandState) + minSpeed;

	tPosd randSpeed;
	randSpeed.x = randSpeedX;
	randSpeed.y = randSpeedY;
	randSpeed.z = randSpeedZ;

	double randAccelX = accelDelta * hiprand_uniform(&hiprandState) + minAccel;
	double randAccelY = accelDelta * hiprand_uniform(&hiprandState) + minAccel;
	double randAccelZ = accelDelta * hiprand_uniform(&hiprandState) + minAccel;

	tPosd randAccel;
	randAccel.x = randAccelX;
	randAccel.y = randAccelY;
	randAccel.z = randAccelZ;

	State xRand = State(randPos, randSpeed, randAccel);

	//------------------------------find parent--------------------------------------

	//the generation didnt work
	if (!ConstraintChecking::validPoint(segArray, nTrackSegs, &xRand, (double) -2.0)){
		return;
	}
		
	State* xNearest = Kernel::nearestNeighbor(&xRand, graph, 2000, actionSimDeltaTime); //GRAPH ITERATOR FUCKUP!
	xRand.setParentGraphIndex(xNearest->getMyGraphIndex());
	printf("nearestIndex: %d\n", xNearest->getMyGraphIndex());

	//------------------------------apply delta--------------------------------------

	DeltaHeuristics::applyDelta(&xRand, xNearest, segArray, nTrackSegs, forwardSegments, neighborDeltaPos, neighborDeltaSpeed);


	double cMin = xNearest->getPathCost() + EvalFunctions::evaluatePathCost(segArray, nTrackSegs, xNearest, &xRand, forwardSegments); //redifine path cost for new coords
	xRand.setPathCost(cMin);

	//printf("parent out!:%f:%f\n", xRand.getPos().x, xRand.getParent()->getPos().x);

	////the delta application didnt work
	if (!ConstraintChecking::validPoint(segArray, nTrackSegs, &xRand, (double) -1.0)){
		return;
	}


	

	//------------------------------push to graph--------------------------------------
	xRand.setMyGraphIndex(offset);
	graph[offset] = xRand;
	


	
}

//This method takes off the CUDA initialization delay on first call during real-time
// because it is done during loading (pre-computing phase)
void Kernel::gpuWarmup(){
	int count;
	hipDeviceProp_t prop;

	hipGetDeviceCount(&count);

	// --- print devices info ----------------------------------------------------------

	for (int i = 0; i<count; i++) {

		hipGetDeviceProperties(&prop, i);

		printf("--- General Information for device %d ---\n", i);
		printf("Name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %d\n", prop.clockRate);
		printf("Device copy overlap: ");
		if (prop.deviceOverlap) printf("Enabled\n");
		else printf("Disabled\n");
		printf("Kernel execution timeout: ");
		if (prop.kernelExecTimeoutEnabled) printf("Enabled\n");
		else printf("Disabled\n");

		printf("--- Memory Information for device %d ---\n", i);
		printf("Total global mem: %ld\n", prop.totalGlobalMem);
		printf("Total constant mem: %ld\n", prop.totalConstMem);
		printf("Max mem pitch: %ld\n", prop.memPitch);
		printf("Texture alignment: %ld\n", prop.textureAlignment);

		printf("--- MP Information for device %d ---\n", i);
		printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp: %d\n", prop.regsPerBlock);
		printf("Threads in warp: %d\n", prop.warpSize);
		printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0],
			prop.maxThreadsDim[1],
			prop.maxThreadsDim[2]);
		printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0],
			prop.maxGridSize[1],
			prop.maxGridSize[2]);

		printf("\n");
	}
	
	// -- create a context -- kernel warmup ---------------------------------------------

	int *f = NULL;
	hipMalloc(&f, sizeof(int));
	warmStart <<< 1, 1 >> >(f);
	hipFree(f);

	hipDeviceSynchronize();
	std::cout << "kernel error: " << hipGetErrorString(hipPeekAtLastError()) << std::endl;


	//-----------------------------------------------------------------------------------

}

State* Kernel::callKernel(tTrackSeg* segArray, int nTrackSegs, State* initialState,
	double minXVertex, double maxXVertex, double minYVertex, double maxYVertex,
	double numIterations,
	int forwardSegments, double neighborDeltaPos, double neighborDeltaSpeed, double actionSimDeltaTime){

	State* auxBestState;

	State* graph = new State[(unsigned int)numIterations+1];
	initialState->setMyGraphIndex(0);
	graph[0] = *initialState;


	State* auxGraph;
	tTrackSeg* auxSegArray;

	double maxPathCost = 0; //just to mock (was not removed as it can still be needed)

	int NUM_BLOCKS = 5;
	int NUM_THREADS_EACH_BLOCK = 100;
	int NUM_THREADS = NUM_BLOCKS*NUM_THREADS_EACH_BLOCK;

	int numPartialIterations = numIterations / NUM_THREADS;

	if (numPartialIterations == 0) numPartialIterations++;


	clock_t mallocTimer;
	clock_t memcpyTimer1;
	clock_t kernelCallTimer;
	clock_t syncronizeTimer;
	clock_t memcpyTimer2;

	mallocTimer = clock();

	hipMalloc(&auxGraph, sizeof(State)*(unsigned int)numIterations);
	hipMalloc(&auxSegArray, sizeof(tTrackSeg)*(unsigned int)nTrackSegs);
	hipMalloc(&auxBestState, sizeof(State));

	mallocTimer = clock() - mallocTimer;
	std::cout << "malloc timer: " << double(mallocTimer) / (double) CLOCKS_PER_SEC << std::endl;

	memcpyTimer1 = clock();

	//hipMemcpy(auxBestPath, bestPath, sizeof(State)*(unsigned int)numIterations, hipMemcpyHostToDevice);
	hipMemcpy(auxSegArray, segArray, sizeof(tTrackSeg)*(unsigned int)nTrackSegs, hipMemcpyHostToDevice);
	hipMemcpy(auxGraph, graph, sizeof(State)*(unsigned int)numIterations, hipMemcpyHostToDevice);

	memcpyTimer1 = clock() - memcpyTimer1;
	std::cout << "memcpy1 timer: " << double(memcpyTimer1) / (double) CLOCKS_PER_SEC << std::endl;

	for (int i = 0; i < numPartialIterations; i++)
	{
		kernelCallTimer = clock();

		CUDAProcedure << < NUM_BLOCKS, NUM_THREADS_EACH_BLOCK >> > (auxSegArray, nTrackSegs, auxGraph, i,
			minXVertex, maxXVertex, minYVertex, maxYVertex,
			NUM_THREADS, maxPathCost, auxBestState,
			forwardSegments, neighborDeltaPos, neighborDeltaSpeed, actionSimDeltaTime);

		kernelCallTimer = clock() - kernelCallTimer;

		syncronizeTimer = clock();

		hipDeviceSynchronize();

		syncronizeTimer = clock() - syncronizeTimer;

		std::cout << "kernell call timer: " << double(kernelCallTimer) / (double) CLOCKS_PER_SEC << std::endl;
		std::cout << "sync timer: " << double(syncronizeTimer) / (double) CLOCKS_PER_SEC << std::endl;

	}
	
	memcpyTimer2 = clock();

	hipMemcpy(graph, auxGraph, sizeof(State)*(unsigned int)numIterations, hipMemcpyDeviceToHost);
	//hipMemcpy(bestState, auxBestState, sizeof(State)*(unsigned int)numIterations, hipMemcpyDeviceToHost);

	memcpyTimer2 = clock() - memcpyTimer2;

	std::cout << "memcpyTimer2 timer: " << double(memcpyTimer2) / (double) CLOCKS_PER_SEC << std::endl;


	hipFree(auxGraph);
	hipFree(auxSegArray);
	hipFree(auxBestState);

	hipDeviceSynchronize();
	std::cout << "kernel error: " << hipGetErrorString(hipPeekAtLastError()) << std::endl;


	

	return graph;
}
