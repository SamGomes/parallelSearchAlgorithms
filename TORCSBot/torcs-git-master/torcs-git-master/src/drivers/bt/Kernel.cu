#include "hip/hip_runtime.h"

#include "Kernel.cuh"


__global__ void CUDAProcedure(tTrackSeg* segArray, int nTrackSegs, State* graph, int stateIterator, 
	double minXVertex, double maxXVertex, double minYVertex, double maxYVertex, 
	int numThreads, double maxPathCost, State* bestPath,
	int forwardSegments, double neighborDeltaPos, double neighborDeltaSpeed, double actionSimDeltaTime){

	int idx = threadIdx.x + blockDim.x*blockIdx.x;

	hiprandState_t hiprandState;


	/* we have to initialize the state */
	hiprand_init(clock(), /* the seed controls the sequence of random values that are produced */
		idx, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&hiprandState);

	int offset = stateIterator*numThreads + idx;

	double trackMapXMin = minXVertex;
	double trackMapXMax = maxXVertex;

	double trackMapXDelta = trackMapXMax - trackMapXMin;

	double trackMapYMin = minYVertex;
	double trackMapYMax = maxYVertex;

	double trackMapYDelta = trackMapYMax - trackMapYMin;


	double trackMapZMin = 0;
	double trackMapZMax = 20;

	double trackMapZDelta = trackMapZMax - trackMapZMin;

	double minSpeed = 0;
	double maxSpeed = 60;

	double speedDelta = maxSpeed - minSpeed;


	double minAccel = 0;
	double maxAccel = 10;

	double accelDelta = maxAccel - minAccel;


	//------------------------generate random point --------------------------------

	double randPosX = trackMapXDelta * hiprand_uniform(&hiprandState) + trackMapXMin;
	double randPosY = trackMapYDelta * hiprand_uniform(&hiprandState) + trackMapYMin;
	double randPosZ = trackMapZDelta * hiprand_uniform(&hiprandState) + trackMapZMin;

	tPosd randPos;
	randPos.x = randPosX;
	randPos.y = randPosY;
	randPos.z = randPosZ;

	double randSpeedX = speedDelta * hiprand_uniform(&hiprandState) + minSpeed;
	double randSpeedY = speedDelta * hiprand_uniform(&hiprandState) + minSpeed;
	double randSpeedZ = speedDelta * hiprand_uniform(&hiprandState) + minSpeed;

	tPosd randSpeed;
	randSpeed.x = randSpeedX;
	randSpeed.y = randSpeedY;
	randSpeed.z = randSpeedZ;

	double randAccelX = accelDelta * hiprand_uniform(&hiprandState) + minAccel;
	double randAccelY = accelDelta * hiprand_uniform(&hiprandState) + minAccel;
	double randAccelZ = accelDelta * hiprand_uniform(&hiprandState) + minAccel;

	tPosd randAccel;
	randAccel.x = randAccelX;
	randAccel.y = randAccelY;
	randAccel.z = randAccelZ;

	State xRand = State(randPos, randSpeed, randAccel);

	//------------------------------find parent--------------------------------------

	//the generation didnt work
	if (!ConstraintChecking::validPoint(segArray, nTrackSegs, &xRand, (double) -2.0)){
		return;
	}
		
	State* xNearest = Kernel::nearestNeighbor(&xRand, graph, 200, actionSimDeltaTime); //GRAPH ITERATOR FUCKUP!
	xRand.setParent(xNearest);

	//------------------------------apply delta--------------------------------------

	DeltaHeuristics::applyDelta(&xRand, xNearest, segArray, nTrackSegs, forwardSegments, neighborDeltaPos, neighborDeltaSpeed);


	double cMin = xNearest->getPathCost() + EvalFunctions::evaluatePathCost(segArray, nTrackSegs, xNearest, &xRand, forwardSegments); //redifine path cost for new coords
	xRand.setPathCost(cMin);

	//printf("parent out!:%f:%f\n", xRand.getPos().x, xRand.getParent()->getPos().x);

	////the delta application didnt work
	if (!ConstraintChecking::validPoint(segArray, nTrackSegs, &xRand, (double) -2.0)){
		return;
	}


	//------------------------------push to graph--------------------------------------
	graph[offset] = xRand;

	//------------------------------check best path--------------------------------------
	

	if (xRand.getPathCost() >= maxPathCost){ //reevaluate best path

		maxPathCost = xRand.getPathCost();
		delete bestPath;
		int pathIndex = 0;
		State* initialNode = &graph[0];
		while (!xRand.getInitialState()){
			//printf("backtracking:%f\n", xRand.getPos().x);
			bestPath[pathIndex] = xRand;
			xRand = *xRand.getParent();
			pathIndex++;
		}
		bestPath[pathIndex] = *initialNode;
	}


	
}


State* Kernel::callKernel(tTrackSeg* segArray, int nTrackSegs,State* initialState,
	double minXVertex, double maxXVertex, double minYVertex, double maxYVertex,
	double numIterations,
	int forwardSegments, double neighborDeltaPos, double neighborDeltaSpeed, double actionSimDeltaTime){

	State* bestPath = new State[(unsigned int)numIterations];
	State* auxBestPath;

	State* graph = new State[(unsigned int)numIterations];
	graph[0] = *initialState;


	State* auxGraph;
	tTrackSeg* auxSegArray;


	double maxPathCost = -1 * DBL_MAX; //force a change


	int NUM_BLOCKS = 2;
	int NUM_THREADS_EACH_BLOCK = 50;
	int NUM_THREADS = NUM_BLOCKS*NUM_THREADS_EACH_BLOCK;

	int numPartialIterations = numIterations / NUM_THREADS;

	if (numPartialIterations == 0) numPartialIterations++;

	//------------------------------ GPU INFO -------------------------------------
	int count;
	hipDeviceProp_t prop;

	hipGetDeviceCount(&count);

	for (int i = 0; i<count; i++) {

		hipGetDeviceProperties(&prop, i);

		printf("--- General Information for device %d ---\n", i);
		printf("Name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %d\n", prop.clockRate);
		printf("Device copy overlap: ");
		if (prop.deviceOverlap) printf("Enabled\n");
		else printf("Disabled\n");
		printf("Kernel execution timeout: ");
		if (prop.kernelExecTimeoutEnabled) printf("Enabled\n");
		else printf("Disabled\n");

		printf("--- Memory Information for device %d ---\n", i);
		printf("Total global mem: %ld\n", prop.totalGlobalMem);
		printf("Total constant mem: %ld\n", prop.totalConstMem);
		printf("Max mem pitch: %ld\n", prop.memPitch);
		printf("Texture alignment: %ld\n", prop.textureAlignment);

		printf("--- MP Information for device %d ---\n", i);
		printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp: %d\n", prop.regsPerBlock);
		printf("Threads in warp: %d\n", prop.warpSize);
		printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0],
			prop.maxThreadsDim[1],
			prop.maxThreadsDim[2]);
		printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0],
			prop.maxGridSize[1],
			prop.maxGridSize[2]);

		printf("\n");
	}

	//-----------------------------------------------------------------------------------

	clock_t mallocTimer;
	clock_t memcpyTimer1;
	clock_t kernelCallTimer;
	clock_t syncronizeTimer;
	clock_t memcpyTimer2;

	mallocTimer = clock();

	hipMalloc(&auxGraph, sizeof(State)*(unsigned int)numIterations);
	hipMalloc(&auxSegArray, sizeof(tTrackSeg)*(unsigned int)nTrackSegs);
	hipMalloc(&auxBestPath, sizeof(State)*(unsigned int)numIterations);

	mallocTimer = clock() - mallocTimer;
	std::cout << "malloc timer: " << double(mallocTimer) / (double) CLOCKS_PER_SEC << std::endl;

	memcpyTimer1 = clock();

	//hipMemcpy(auxBestPath, bestPath, sizeof(State)*(unsigned int)numIterations, hipMemcpyHostToDevice);
	hipMemcpy(auxSegArray, segArray, sizeof(tTrackSeg)*(unsigned int)nTrackSegs, hipMemcpyHostToDevice);
	hipMemcpy(auxGraph, graph, sizeof(State)*(unsigned int)numIterations, hipMemcpyHostToDevice);

	memcpyTimer1 = clock() - memcpyTimer1;
	std::cout << "memcpy1 timer: " << double(memcpyTimer1) / (double) CLOCKS_PER_SEC << std::endl;

	for (int i = 0; i < numPartialIterations; i++)
	{
		kernelCallTimer = clock();

		CUDAProcedure << < NUM_BLOCKS, NUM_THREADS_EACH_BLOCK >> > (auxSegArray, nTrackSegs, auxGraph, i,
			minXVertex, maxXVertex, minYVertex, maxYVertex,
			NUM_THREADS, maxPathCost, auxBestPath,
			forwardSegments, neighborDeltaPos, neighborDeltaSpeed, actionSimDeltaTime);

		kernelCallTimer = clock() - kernelCallTimer;

		syncronizeTimer = clock();

		hipDeviceSynchronize();

		syncronizeTimer = clock() - syncronizeTimer;

		std::cout << "kernell call timer: " << double(kernelCallTimer) / (double) CLOCKS_PER_SEC << std::endl;
		std::cout << "sync timer: " << double(syncronizeTimer) / (double) CLOCKS_PER_SEC << std::endl;

	}
	
	memcpyTimer2 = clock();

	hipMemcpy(graph, auxGraph, sizeof(State)*(unsigned int)numIterations, hipMemcpyDeviceToHost);
	hipMemcpy(bestPath, auxBestPath, sizeof(State)*(unsigned int)numIterations, hipMemcpyDeviceToHost);

	memcpyTimer2 = clock() - memcpyTimer2;

	std::cout << "memcpyTimer2 timer: " << double(memcpyTimer2) / (double) CLOCKS_PER_SEC << std::endl;


	//hipDeviceSynchronize();

	/*for (int i = 0; i < 10; i++){
		printf("dgraphPos!:%f\n", graph[i].getPos().x);
	}*/

	hipFree(auxGraph);
	hipFree(auxSegArray);
	hipFree(auxBestPath);

	std::cout << "gcnsjfddgjvhgdffgshgfuiserror: " << hipGetErrorString(hipPeekAtLastError()) << std::endl;


	

	delete[] graph;

	return bestPath;
}
